
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define SIZE 1000

__global__ void helloKernel() {
    printf("Hello, CUDA!\n");
}


int main() {

	
  helloKernel<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
