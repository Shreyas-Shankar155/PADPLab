
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 10

__global__ void matrixAdd(int *A, int *B, int *C){
	
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int col = threadIdx.y + blockIdx.y * blockDim.y;
	
	//global index = row * max_num_of_cols + col
	int idx = row*SIZE + col; //column max size is hardcoded here
	
	C[idx] = A[idx] + B[idx];		
	
	}

void arr_init(int arr[SIZE][SIZE]){
	
	for(int i=0; i<SIZE; i++)
		for(int j=0; j<SIZE; j++)
			arr[i][j] = rand()%10;
}

void print_arr(int arr[SIZE][SIZE], char ch){
	printf("Array %c:\n",ch);
	for(int i=0; i<SIZE; i++)
		{
		for(int j=0; j<SIZE; j++)
			printf("%d\t",arr[i][j]);
		printf("\n");
		}
}
int main(){
	int A[SIZE][SIZE];
	int B[SIZE][SIZE];
	int C[SIZE][SIZE];
	arr_init(A); print_arr(A, 'A');
	arr_init(B); print_arr(B, 'B');
	
	int *arr_a, *arr_b, *arr_c;
	hipMalloc(&arr_a, sizeof(A));
	hipMalloc(&arr_b, sizeof(B));
	hipMalloc(&arr_c, sizeof(C));
	
	hipMemcpy(arr_a, A, sizeof(A), hipMemcpyHostToDevice);
	hipMemcpy(arr_b, B, sizeof(B), hipMemcpyHostToDevice);
	
	dim3 blockDim(16,16); //create thread block of 16*16 threads
	
	matrixAdd<<<1,blockDim>>> (arr_a, arr_b, arr_c);
	hipMemcpy(C, arr_c, sizeof(C), hipMemcpyDeviceToHost);
	
	print_arr(C,'C');
	hipFree(arr_a);
    	hipFree(arr_b);
   	hipFree(arr_c);
	
	return 0;
}
