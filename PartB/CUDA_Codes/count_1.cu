
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define SIZE 1000
/*
__global__ void helloKernel() {
    printf("Hello, CUDA!\n");
}
*/

__global__ void count1s(int *arr, int *res)
	{
	__shared__ int x;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(threadIdx.x==0)
		x = 0;
	__syncthreads();
	
	if(tid<SIZE && arr[tid]==1)
		atomicAdd(&x,1);
	__syncthreads();
	
	if (threadIdx.x == 0) {
        	atomicAdd(res, x);
    		}
	}

int main() {

	int myRes = 0;
	int A[SIZE];
	for(int i=0; i<SIZE; i++)
		myRes += (A[i] = rand()%2)==1?1:0;
	printf("Number of 1s from CPU: %d\n",myRes);	
  	myRes=0;  
  	
    int *res;
    int *arr;
    
    hipMalloc(&arr, sizeof(A));
    
    hipMalloc(&res, sizeof(int));
    hipMemset(res, 0, sizeof(int));
    
    hipMemcpy(arr, A, sizeof(A), hipMemcpyHostToDevice);
    
    int blocks = (int)(SIZE/64)+1;
    count1s<<<blocks,64>>> (arr, res);
    
    hipDeviceSynchronize();
    
    hipMemcpy(&myRes, res, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Number of 1s from GPU: %d\n",myRes);
    return 0;
}
