/*
Counts number of 1s in an array
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 100000

__global__ void count1s(int *arr, int *res)
	{
	__shared__ int val; //creates shared memory for all threads in thread block
	int tid = threadIdx.x + blockIdx.x * blockDim.x; //find index according to array
	
	if(threadIdx.x==0)
		val = 0; //per block, assign one thread to value 0
	__syncthreads();
	
	if(tid<SIZE && arr[tid]==1) //per block, keep sum of number of 1s
		atomicAdd(&val,1); 
	__syncthreads();
	
	if (threadIdx.x == 0) { //master thread per block updates result variable
        	atomicAdd(res, val);
    		}
	}

int main() {

	int myRes = 0;
	int A[SIZE];
	for(int i=0; i<SIZE; i++)
		myRes += (A[i] = rand()%2)==1?1:0;
	printf("Number of 1s from CPU: %d\n",myRes);	
  	myRes=0;  
  	
    int *res; //stores result
    int *arr; //stores array A
        
    hipMalloc(&res, sizeof(int));
    hipMemset(res, 0, sizeof(int));
    
    hipMalloc(&arr, sizeof(A));
    hipMemcpy(arr, A, sizeof(A), hipMemcpyHostToDevice);
    
    int blocks = (int)(SIZE/64)+1;
    count1s<<<blocks,64>>> (arr, res);
    
//    cudaDeviceSynchronize();
    
    hipMemcpy(&myRes, res, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Number of 1s from GPU: %d\n",myRes);
    return 0;
}
